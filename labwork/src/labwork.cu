#include "hip/hip_runtime.h"
#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <math.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTH ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg"); 
            printf("labwork 1 OpenMP ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
            break;
        case 5:
            labwork.labwork5_CPU();
            labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}
void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    // do something here
    #pragma omp parallel for
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
            	                        (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
            // if(outputImage[i*3] < 0) printf("get negative ! \n");
        }
    }
}

int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int nDevices = 0;
    // get all devices
    hipGetDeviceCount(&nDevices);
    printf("Number total of GPU : %d\n\n", nDevices);
    for (int i = 0; i < nDevices; i++){
        // get informations from individual device
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        // something more here
	printf("Device Number: %d\n", i);
	printf("  Device Name: %s\n", prop.name);
	printf("  CoreInfor :) \n    Clock Rate(KHz): %d\n    Core Count: %d\n    Multiple Processor Count: %d\n    Warp Size: %d\n", prop.clockRate, getSPcores(prop), prop.multiProcessorCount, prop.warpSize);
    	printf("  MemoryInfor :[] \n    Clock Rate(KHz): %d\n    Bus Width(bits): %d\n    Band Width(GB/s): %f\n", prop.memoryClockRate, prop.memoryBusWidth, 2.0 * prop.memoryClockRate * (prop.memoryBusWidth/8)/1.0e6);
    }

}
__global__ void grayscale(uchar3* input, uchar3* output){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	output[tid].x = (input[tid].x + input[tid].y + input[tid].z) / 3;
	output[tid].z = output[tid].y = output[tid].x;
}

void Labwork::labwork3_GPU() {
    // Calculate number of pixels
    int pixelCount = inputImage->width * inputImage->height;
    // Allocate host output memory
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    // Allocate CUDA memory    
    uchar3* dev_input;
    uchar3* dev_output;
    hipMalloc(&dev_input, pixelCount * sizeof(uchar3));
    hipMalloc(&dev_output, pixelCount * sizeof(uchar3));
    // Copy CUDA Memory from CPU to GPU
    hipMemcpy(dev_input, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);
    // Define blockSize and numBlock
    int blockSize = 64;
    int numBlock = pixelCount / blockSize;
    // Processing
    grayscale<<<numBlock, blockSize>>>(dev_input, dev_output);
    // Copy CUDA Memory from GPU to CPU
    hipMemcpy(outputImage, dev_output, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost);
    // Cleaning
    hipFree(dev_input);
    hipFree(dev_output);
}

__global__ void grayscaleVer2D(uchar3* input, uchar3* output, int width, int height){
	int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
	int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
	if(tid_x > width || tid_y > height) return;
	int tid = (int)(tid_x + tid_y * width);
	output[tid].x = (input[tid].x + input[tid].y + input[tid].z) / 3;
	output[tid].z = output[tid].y = output[tid].x;
}

void Labwork::labwork4_GPU() {
	int pixelCount = inputImage->width * inputImage->height;
	outputImage = static_cast<char*>(malloc(pixelCount * 3));
	uchar3* dev_input;
        uchar3*	dev_output;
	hipMalloc(&dev_input, pixelCount * sizeof(uchar3));
	hipMalloc(&dev_output, pixelCount * sizeof(uchar3));
	hipMemcpy(dev_input, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);
	// set value for block and grid
	int b_x = 32;
	int b_y = 32;
	int d_x = (int)(inputImage->width / b_x) + (inputImage->width % b_x == 0?0:1);
	int d_y = (int)(inputImage->height / b_y) + (inputImage->height % b_y == 0?0:1);
	// execute processing
	dim3 blockSize = dim3(b_x, b_y);
	dim3 gridSize = dim3(d_x, d_y);
	grayscaleVer2D<<<gridSize, blockSize>>>(dev_input, dev_output, inputImage->width, inputImage->height);
	hipMemcpy(outputImage, dev_output, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost);
	hipFree(dev_input);
	hipFree(dev_output);
}


__global__ void blur_convol2D (uchar3* dev_input, uchar3* dev_output, int width, int height){
	//int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
	//int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
	//int tid_z = threadIdx.z + blockIdx.z * blockDim.z;
	//if (tid_x > width || tid_y > height) return;
	//__shared__ char shared_image[];
	//shared_image[tid] = input[tid];
	//__syncthreads();

}

void Labwork::labwork5_GPU() {

	int pixelCount = inputImage->width * inputImage->height;
	outputImage = static_cast<char*>(malloc(pixelCount * 3));
	uchar3* dev_input;
        uchar3*	dev_output;
	hipMalloc(&dev_input, pixelCount * sizeof(uchar3));
	hipMalloc(&dev_output, pixelCount * sizeof(uchar3));
	hipMemcpy(dev_input, inputImage->buffer, pixelCount * sizeof(uchar3), hipMemcpyHostToDevice);
	// set value for block and grid
	int b_x = 32;
	int b_y = 32;
	int d_x = (int)(inputImage->width / b_x) + 1;
	int d_y = (int)(inputImage->height / b_y) + 1;
	// execute processing
	dim3 blockSize = dim3(b_x, b_y);
	dim3 gridSize = dim3(d_x, d_y);
	blur_convol2D<<<gridSize, blockSize>>>(dev_input, dev_output, inputImage->width, inputImage->height);
	hipMemcpy(outputImage, dev_output, pixelCount * sizeof(uchar3), hipMemcpyDeviceToHost);
	hipFree(dev_input);
	hipFree(dev_output);
}

void Labwork::labwork5_CPU() {
}

void Labwork::labwork6_GPU() {
}

void Labwork::labwork7_GPU() {
}

void Labwork::labwork8_GPU() {
}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU(){
}


























